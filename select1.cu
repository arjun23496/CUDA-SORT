
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__device__ void swap(int &a,int &b)
{
	int c;
	c=a;
	a=b;
	b=c;
}	

__device__ int partition(int *array,int l,int u)
{
	int p=array[u-1];
	int j=l-1;

	for(int i=l;i<u-1;i++)
	{
		if(array[i]<p)
		{
			j++;

			swap(array[j],array[i]);			
		}
	}
	j++;
	swap(array[j],array[u-1]);

	return j;
}

__device__ int quickSelect(int *array,int l,int u,int x)
{
	int q=array[l];
	if(l<u)
	{
		q=partition(array,l,u);
		
		if(x<q)
		{
			return quickSelect(array,l,q,x);
		}
		else if(x>q)
		{	
			return quickSelect(array,q+1,u,x);
		}
		else
		{	
			return array[q];	
		}

	}
	return array[q];
}

__global__ void findMedian(int* medians,int* d_in,int n,int N,int k){

	int idx=threadIdx.x;
	int multiplier=ceil((double)n/N);
	int l=multiplier*idx;
	int u=multiplier*(idx+1);
	int m=l+((u-l)/2);

	u=(u>n)?n:u;

	medians[idx]=quickSelect(d_in,l,u,m);

}

void swapHost(int &a,int &b)
{
	int c;
	c=a;
	a=b;
	b=c;
}	

int partitionHost(int *array,int l,int u)
{
	int p=array[u-1];
	int j=l-1;

	for(int i=l;i<u-1;i++)
	{
		if(array[i]<p)
		{
			j++;

			swapHost(array[j],array[i]);			
		}
	}
	j++;
	swapHost(array[j],array[u-1]);

	return j;
}

int quickSelectHost(int *array,int l,int u,int x)
{
	int q=array[l];
	if(l<u)
	{
		q=partitionHost(array,l,u);
		
		if(x<q)
		{
			return quickSelectHost(array,l,q,x);
		}
		else if(x>q)
		{	
			return quickSelectHost(array,q+1,u,x);
		}
		else
		{	
			return array[q];	
		}

	}
	return array[q];
}

int main(){

	const int ARRAY_SIZE_INPUT=100000;
	const int ARRAY_BYTES_INPUT= ARRAY_SIZE_INPUT*sizeof(int);
	const int NUMBER_OF_PROCESSORS=1000;
	const int MEDIAN_BYTES=NUMBER_OF_PROCESSORS*sizeof(int);
	int k=997;

	int ARRAY_SIZE=ARRAY_SIZE_INPUT;
	int ARRAY_BYTES=ARRAY_BYTES_INPUT;
	int L[ARRAY_SIZE];
	int E[ARRAY_SIZE];
	int G[ARRAY_SIZE];
	int result=0;

	//generate input array
	int array_in[ARRAY_SIZE];
	int initial_array[ARRAY_SIZE];
	
	srand(time(NULL));

	//Array of random integers TODO: srand()
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		array_in[i]=rand()%10000;
		initial_array[i]=array_in[i];
	}

	printf("Input-------------------------------------------\n");
	// Array of random numbers
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		printf("%d ",array_in[i]);
	}
	printf(" k value %d\n",k );
	printf("\n");

	while(true)
	{

		if(ARRAY_SIZE<NUMBER_OF_PROCESSORS)
		{
			result=quickSelectHost(array_in,0,ARRAY_SIZE,k);
			break;
		}

		//cuda variables
		int *d_in;
		int *medians;

		//host medians
		int host_medians[NUMBER_OF_PROCESSORS];
		int device_array[ARRAY_SIZE];

		hipMalloc((void**)&d_in,ARRAY_BYTES);
		hipMalloc((void**)&medians,MEDIAN_BYTES);

		hipMemcpy(d_in,array_in,ARRAY_BYTES,hipMemcpyHostToDevice);

		findMedian<<<1,NUMBER_OF_PROCESSORS>>>(medians,d_in,ARRAY_SIZE,NUMBER_OF_PROCESSORS,k);

		hipMemcpy(host_medians,medians,MEDIAN_BYTES,hipMemcpyDeviceToHost);
		hipMemcpy(device_array,d_in,ARRAY_BYTES,hipMemcpyDeviceToHost);

		printf("Medians------------------------------------------------------------\n");

		for(int i=0;i<NUMBER_OF_PROCESSORS;i++)
		{
			printf("%d\n",host_medians[i]);
		}

	printf("\n");
	printf("device array------------------------------------------------------------\n");

	for(int i=0;i<ARRAY_SIZE;i++)
	{
		printf("%d ",device_array[i]);
	}


		printf("\nMedian of medians------------------------------------------------------------\n");

		int medianOfMedians=quickSelectHost(host_medians,0,NUMBER_OF_PROCESSORS,NUMBER_OF_PROCESSORS/2);	

		printf("\n%d\n",medianOfMedians);


		//Clasification

		int lctr=0;
		int ectr=0;
		int gctr=0;

		for(int i=0;i<ARRAY_SIZE;i++)
		{
			if(array_in[i]<medianOfMedians)
			{
				L[lctr]=array_in[i];
				lctr++;
			}
			else
			{
				if(array_in[i]>medianOfMedians)
				{
					G[gctr]=array_in[i];
					gctr++;
				}
				else
				{
					E[ectr]=array_in[i];
					ectr++;
				}
			}
		}

		printf("\n");
		printf("Lesser........................................................");
		printf("\n");

		for(int i=0;i<lctr;i++)
		{
			printf("%d ",L[i]);
		}


		printf("\n");
		printf("Equal........................................................");
		printf("\n");

		for(int i=0;i<ectr;i++)
		{
			printf("%d ",E[i]);
		}


		printf("\n");
		printf("Greater........................................................");
		printf("\n");

		for(int i=0;i<gctr;i++)
		{
			printf("%d ",G[i]);
		}
		printf("\n");

		// Check for completion
		// int u=ARRAY_SIZE;
		if(lctr>=k)
		{
			for(int i=0;i<lctr;i++)
			{
				array_in[i]=L[i];
				ARRAY_SIZE=lctr;
				ARRAY_BYTES=ARRAY_SIZE*sizeof(int);
			}
		}
		else 
		{
			if(lctr+ectr>=k)
			{
				// l=lctr;
				// u=lctr;
				result=E[0];
				break;
			}
			else
			{
				k=k-(lctr+ectr);					
				for(int i=0;i<gctr;i++)
				{
					array_in[i]=G[i];
					ARRAY_SIZE=gctr;
					ARRAY_BYTES=ARRAY_SIZE*sizeof(int);
				}
			}
		}

		hipFree(d_in);
		hipFree(medians);
	}

	printf("\nInitial Input-------------------------------------------\n");
	// Array of random numbers
	for(int i=0;i<ARRAY_SIZE_INPUT;i++)
	{
		printf("%d ",initial_array[i]);
	}
	printf(" k value %d\n",k );
	printf("\n");

	printf("Result is %d\n",result);	

	return 0;

}