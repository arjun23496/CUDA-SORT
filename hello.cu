
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void square(float* d_out,float* d_in){
	int idx= threadIdx.x;
	float f= d_in[idx];
	d_out[idx]=f*f;
	// d_out[idx]=idx;
}

int main(){
	const int ARRAY_SIZE=64;
	const int ARRAY_BYTES= ARRAY_SIZE*sizeof(float);

	//generate input array
	float h_in[ARRAY_SIZE];
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		h_in[i]=float(i);
	}
	float h_out[ARRAY_SIZE];

	float *d_in;
	float *d_out;

	hipMalloc((void**)&d_in,ARRAY_BYTES);
	hipMalloc((void**)&d_out,ARRAY_BYTES);

	hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice);

	square<<<1,ARRAY_SIZE>>>(d_out,d_in);

	hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost);

	printf("\n");

	for(int i=0;i<ARRAY_SIZE;i++)
	{
		printf("%f\n",h_out[i]);
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;

}
