
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void swapHost(int &a,int &b)
{
	int c;
	c=a;
	a=b;
	b=c;
}	

int partitionHost(int *array,int l,int u)
{
	int p=array[u-1];
	int j=l-1;

	for(int i=l;i<u-1;i++)
	{
		if(array[i]<p)
		{
			j++;

			swapHost(array[j],array[i]);			
		}
	}
	j++;
	swapHost(array[j],array[u-1]);

	return j;
}

int quickSelectHost(int *array,int l,int u,int x)
{
	int q=array[l];
	if(l<u)
	{
		q=partitionHost(array,l,u);
		
		if(x<q)
		{
			return quickSelectHost(array,l,q,x);
		}
		else if(x>q)
		{	
			return quickSelectHost(array,q+1,u,x);
		}
		else
		{	
			return array[q];	
		}

	}
	return array[q];
}

int cmpfunc (const void * a, const void * b)
{
   return ( *(int*)a - *(int*)b );
}


int main(){

	const int ARRAY_SIZE_INPUT=100000;
	// const int ARRAY_BYTES_INPUT= ARRAY_SIZE_INPUT*sizeof(int);
	int k=5000;

	int ARRAY_SIZE=ARRAY_SIZE_INPUT;
	int result=0;

	//generate input array
	int array_in[ARRAY_SIZE];
	
	// srand(time(NULL));

	//Array of random integers TODO: srand()
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		array_in[i]=rand()%10000;
	}

	printf("Input-------------------------------------------\n");
	// Array of random numbers
	for(int i=0;i<ARRAY_SIZE;i++)
	{
		printf("%d ",array_in[i]);
	}
	printf(" k value %d\n",k );
	printf("\n");

	// result=quickSelectHost(array_in,0,ARRAY_SIZE,k);

	qsort(array_in,ARRAY_SIZE,sizeof(int),cmpfunc);

	result=array_in[k];

	printf(" result is := %d\n",result);

	return 0;

}