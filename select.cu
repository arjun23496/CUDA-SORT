#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
// #include <hiprand/hiprand_kernel.h>

__device__ int quickSelect(int* array,int l,int u,int index)
{
	int n=u-l;
	int l1=l+1;
	int u1=u-1;
	int i=0;
	int pivot=l;

	if(n==1)
	{
		return array[l];
		// return l;
	}

	while(l1<u1&&i<1000)
	{
		i++;

		if(array[l1]>array[pivot] && array[u1]<array[pivot])
		{
			//swap
			array[l1]+=array[u1];
			array[u1]=array[l1]-array[u1];
			array[l1]=array[l1]-array[u1];
			l1++;		
		}

		if(array[l1]<=array[pivot])
		{
			l1++;
		}
		
		if(array[u1]>array[pivot])
		{
			u1--;
		}
	}

	int pivotEle=array[pivot];
	int pivotPos=l;

	// return 0;

	for(i=l+1;pivotEle>=array[i]&&i<u;i++)
	{
		array[i-1]=array[i];
	}

	array[i-1]=pivotEle;
	
	pivotPos=i-1;
	
	if(pivotPos==index)
	{
		return array[pivotPos];
		// return 555;
	}

	int e;

	if(pivotPos>index)
	{
		e=quickSelect(array,l,pivotPos,index);
		// e=22;
		return e;
	}
	else
	{
		e=quickSelect(array,pivotPos+1,u,index);
		// e=11;
		return e;
	}
}

__global__ void findMedian(int* medians,int* d_in,int n,int N){

	int multiplier=ceil((double)n/N);
	int idx=threadIdx.x;
	int l=idx*multiplier;
	int u=(idx+1)*multiplier;
	int m=l;

	u=(u>n)?n:u;
	m=l+((u-l)/2);

	// medians[idx]=quickSelect(d_in,l,u,m);
	thrust::sort(thrust::seq, d_in+l, d_in+u);
	medians[idx]=d_in[m];
}

int quickSelectHost(int* array,int l,int u,int index)
{
	int n=u-l;
	int l1=l+1;
	int u1=u-1;
	int i=0;
	int pivot=l;

	if(n<=1)
	{
		return array[l];
		// return l;
	}

	while(l1<u1&&i<1000)
	{
		i++;

		if(array[l1]>array[pivot] && array[u1]<array[pivot])
		{
			//swap
			array[l1]+=array[u1];
			array[u1]=array[l1]-array[u1];
			array[l1]=array[l1]-array[u1];
			l1++;		
		}

		if(array[l1]<=array[pivot])
		{
			l1++;
		}
		
		if(array[u1]>array[pivot])
		{
			u1--;
		}
	}

	int pivotEle=array[pivot];
	int pivotPos=l;

	// return 0;

	for(i=l+1;pivotEle>=array[i]&&i<u;i++)
	{
		array[i-1]=array[i];
	}

	array[i-1]=pivotEle;
	
	pivotPos=i-1;
	
	if(pivotPos==index)
	{
		return array[pivotPos];
		// return 555;
	}

	int e;

	if(pivotPos>index)
	{
		e=quickSelectHost(array,l,pivotPos,index);
		// e=22;
		return e;
	}
	else
	{
		e=quickSelectHost(array,pivotPos+1,u,index);
		// e=11;
		return e;
	}
}

int cmpfunc (const void * a, const void * b)
{
   return ( *(int*)a - *(int*)b );
}

int main(){

	const int ARRAY_SIZE_INPUT=13;
	const int ARRAY_BYTES_INPUT = ARRAY_SIZE_INPUT*sizeof(int);
	const int NUMBER_OF_PROCESSORS=10;
	int k=7;

	int ARRAY_SIZE=ARRAY_SIZE_INPUT;
	int ARRAY_BYTES=ARRAY_BYTES_INPUT;
	int MEDIAN_BYTES=NUMBER_OF_PROCESSORS*sizeof(int);	
	// int array_in[ARRAY_SIZE];
	// int array_in_copy[ARRAY_SIZE];
	int medianOfMedians=0;
	int L[ARRAY_SIZE];
	int E[ARRAY_SIZE];
	int G[ARRAY_SIZE];
	int result=0;

	//Cuda pointers
	int *d_in=NULL;
	int *medians=NULL;
	
	//Random number seed
	srand(time(NULL));

	int array_in[]={4, 2, 6, 4 ,4, 2, 10, 13, 0, 13, 4, 13, 9};

	// Array of random numbers
	// for(int i=0;i<ARRAY_SIZE;i++)
	// {
	// 	array_in[i]=rand()%20;
	// 	// array_in_copy[i]=array_in[i];
	// }

	printf("\n");

	while(true)
	{
		printf("---------------------------Ieration ------------------------------------");
		printf("\n");
		for(int i=0;i<ARRAY_SIZE;i++)
		{
			printf("%d ",array_in[i]);
			// array_in_copy[i]=array_in[i];
		}
		printf("\n");
		printf("k value is %d \n",k);
		printf("\n");		

		if(ARRAY_SIZE<NUMBER_OF_PROCESSORS)
		{
			qsort(array_in, ARRAY_SIZE, sizeof(int), cmpfunc);
			result=array_in[k];
			break;
		}

		int host_median[NUMBER_OF_PROCESSORS];

		//Allocate cuda device memory
		hipMalloc((void**)&d_in,ARRAY_BYTES);
		hipMalloc((void**)&medians,MEDIAN_BYTES);
		
		//Copy input array to device
		hipMemcpy(d_in,array_in,ARRAY_BYTES,hipMemcpyHostToDevice);

		//Find Medians
		findMedian<<<1,NUMBER_OF_PROCESSORS>>>(medians,d_in,ARRAY_SIZE,NUMBER_OF_PROCESSORS);

		hipMemcpy(host_median,medians,MEDIAN_BYTES,hipMemcpyDeviceToHost);

		printf("---------------------Medians-----------------------------------\n");

		for(int i=0;i<NUMBER_OF_PROCESSORS;i++)
		{
			printf("%d\n",host_median[i]);
		}

		printf("---------------------Median of medians-----------------------------------\n");

		medianOfMedians=quickSelectHost(host_median,0,NUMBER_OF_PROCESSORS,NUMBER_OF_PROCESSORS/2);

		printf("%d\n",medianOfMedians);

		//Classification

		int lctr=0;
		int ectr=0;
		int gctr=0;

		for(int i=0;i<ARRAY_SIZE;i++)
		{
			if(array_in[i]<medianOfMedians)
			{
				L[lctr]=array_in[i];
				lctr++;
			}
			else
			{
				if(array_in[i]>medianOfMedians)
				{
					G[gctr]=array_in[i];
					gctr++;
				}
				else
				{
					E[ectr]=array_in[i];
					ectr++;
				}
			}
		}

		printf("\n");
		printf("Lesser........................................................");
		printf("\n");

		for(int i=0;i<lctr;i++)
		{
			printf("%d ",L[i]);
		}


		printf("\n");
		printf("Equal........................................................");
		printf("\n");

		for(int i=0;i<ectr;i++)
		{
			printf("%d ",E[i]);
		}


		printf("\n");
		printf("Greater........................................................");
		printf("\n");

		for(int i=0;i<gctr;i++)
		{
			printf("%d ",G[i]);
		}
		printf("\n");

		// Check for completion
		// int u=ARRAY_SIZE;
		if(lctr>=k)
		{
			for(int i=0;i<lctr;i++)
			{
				array_in[i]=L[i];
				ARRAY_SIZE=lctr;
				ARRAY_BYTES=ARRAY_SIZE*sizeof(int);
			}
		}
		else 
		{
			if(lctr+ectr>=k)
			{
				// l=lctr;
				// u=lctr;
				result=E[0];
				break;
			}
			else
			{
				k=k-(lctr+ectr);					
				for(int i=0;i<gctr;i++)
				{
					array_in[i]=G[i];
					ARRAY_SIZE=gctr;
					ARRAY_BYTES=ARRAY_SIZE*sizeof(int);
				}
			}
		}


		//Free Cuda memory
		hipFree(d_in);
		hipFree(medians);
	}

	printf("Result is %d \n",result );

	return 0;

}